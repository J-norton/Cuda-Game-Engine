#include "hip/hip_runtime.h"

#include "CudaGLBuffer.cuh"

__global__ void simple_vbo_kernel(float *pos, int *indices, int num_vertices, int timestep)
{
	unsigned int x = blockIdx.x * timestep;
	int stride = 8;

	int idx_1 = indices[x] * stride;
	int idx_2 = indices[x + 1] * stride;
	int idx_3 = indices[x + 2] * stride;

	float4 factor = float4();
	float mul = 0.9;
	for (int i = 0; i < 3; i++)
	{
		pos[idx_1] = pos[idx_1] * mul;
		pos[idx_2] = pos[idx_2] * mul;
		pos[idx_3] = pos[idx_3] * mul;

		idx_1 += 1;
		idx_2 += 1;
		idx_3 += 1;
	}
}

CudaGLBufferObj::CudaGLBufferObj(void)
{
}

CudaGLBufferObj::~CudaGLBufferObj(void)
{
}

std::string CudaGLBufferObj::getType()
{
	return "CUDA + GL VBO";
}

void CudaGLBufferObj::bind()
{
	//If the parameter size is zero, there is no data to bind to the GL
	if (parameters.size() == 0) {
		std::cout << "Empty buffers for shader object: \n";
		return;
	}

	if (!bound) {
		initBuffers();
		bound = true;
	}
	else
		runCuda();

	glBindVertexArray(vao_handle);
	if (has_indices)
		bindIndices();
}

void CudaGLBufferObj::runCuda()
{
	hipError_t error_code = hipSuccess;
	error_code = hipGraphicsMapResources(1, &cuda_vbo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL buffer" << std::endl;

	size_t num_bytes;
	size_t num_bytes_ibo;
	error_code = hipGraphicsResourceGetMappedPointer(&dev_arg1, &num_bytes, cuda_vbo);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL buffer" << std::endl;

	error_code = hipGraphicsMapResources(1, &cuda_ibo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL index buffer" << std::endl;

	error_code = hipGraphicsResourceGetMappedPointer(&dev_indices, &num_bytes_ibo, cuda_ibo);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL buffer" << std::endl;

	int timestep;
	launchKernel();
	error_code = hipGraphicsUnmapResources(1, &cuda_vbo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error unmapping GL buffer" << std::endl;
	error_code = hipGraphicsUnmapResources(1, &cuda_ibo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error unmapping GL index buffer" << std::endl;
}

void CudaGLBufferObj::initIndexBuffer(int* indices, int n)
{
	has_indices = true;
	unsigned int* temp = new unsigned int[1];
	num_vertices = n / 4;

	glGenBuffers(1, temp);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, temp[0]);
	index_handle = temp[0];
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, n, indices,
		GL_STATIC_DRAW);
	// Unbind buffer
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

	hipError_t error = hipGraphicsGLRegisterBuffer(&cuda_ibo, index_handle, cudaGraphicsMapFlagsWriteDiscard);

	if (error != hipSuccess)
		std::cout << "Error occurred mapping index buffer to CUDA" << std::endl;

	//Now that the memory is buffered, delete the client-side ints
	delete[] indices;
}

void CudaGLBufferObj::launchKernel()
{
	dim3 block(16, 1, 1);
	dim3 grid(1, 1, 1);
	simple_vbo_kernel<<<block, grid>>>((float*)dev_arg1, (int*) dev_indices, num_vertices, timestep++);
}

void CudaGLBufferObj::unbind()
{
	glBindVertexArray(0);
}

void CudaGLBufferObj::initBuffers()
{
	timestep = 0;
	//First determine the size of the entire buffer
	int buffer_size = 0;
	//Represents the number of bytes between the ith and i + 1th item in the buffer
	stride = 0;
	for (ShaderParameter p : parameters)
	{
		buffer_size += p.size;
		stride += p.cardinality * 4;
		p.shader_handle = WorldState::getCurrentShader()->getAttrib(p.name);
	}

	int vertex_count = buffer_size / stride;
	float* interleaved_data = new float[buffer_size / 4];
	int k = 0;

	//The following assumes that all parameters have the same number of elements
	for (int i = 0; i < vertex_count; i++)
	{
		for (ShaderParameter p : parameters)
		{
			//The index into the array that will
			int offset = p.cardinality * i;
			for (int j = 0; j < p.cardinality; j++)
				interleaved_data[k++] = (p.client_data[offset + j]);
		}
	}
	//Create a Vertex Array Object so that we can bind the VBOs to it
	glGenVertexArrays(1, &vao_handle);
	glBindVertexArray(vao_handle);
	GLuint* buffer_return = new GLuint[1];
	//Finally generate and bind the buffer
	glGenBuffers(1, buffer_return);
	interleave_handle = buffer_return[0];
	glBindBuffer(GL_ARRAY_BUFFER, interleave_handle);
	glBufferData(GL_ARRAY_BUFFER, buffer_size, &interleaved_data[0], GL_STATIC_DRAW);
	//Describe the layout of the data to the GL
	GLvoid* offset = 0;
	int address = 0;
	for (ShaderParameter parameter : parameters)
	{
		//GLuint shader_num = WorldState::current_shader->getAttrib(parameter.name);
		GLuint shader_num = WorldState::getCurrentShader()->getAttrib(parameter.name);
		if (shader_num == -1)
		{
			address += parameter.cardinality * 4;
			offset = (GLvoid*) address;
			continue;
		}
		glVertexAttribPointer(shader_num, parameter.cardinality, GL_FLOAT, false, stride, offset);
		if (glGetError() != 0)
			std::cout << glGetError() << std::endl << "Illegal arguments set for framebuffer texture";

		address += parameter.cardinality * 4;
		offset = (GLvoid*) address;
		glEnableVertexAttribArray(shader_num);
		if (glGetError() != 0)
			std::cout << glGetError() << std::endl << "Illegal arguments set for framebuffer texture";
	}
	//Unbind
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	hipError_t err = hipGraphicsGLRegisterBuffer(&cuda_vbo, interleave_handle, cudaGraphicsMapFlagsWriteDiscard);
	if (err != 0)
		std::cout << "Could not map GL Buffer to CUDA" << std::endl;

	delete[] buffer_return;
	delete[] interleaved_data;
	//Delete all individual client-side arrays
	for (ShaderParameter p : parameters)
	{
		if (p.delete_on_buffer)
			delete[] p.client_data;
	}
	bound = true;
}