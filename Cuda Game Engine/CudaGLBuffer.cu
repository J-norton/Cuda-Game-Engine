#include "hip/hip_runtime.h"

#include "CudaGLBuffer.cuh"


__device__ float3 operator+(const float3 &a, const float3 &b)
{
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator* (const float3 &a, const float &b)
{
	return make_float3(a.x * b, a.y * b, a.z * b);
}

__global__ void simple_vbo_kernel(float *pos, int *indices, int num_vertices, int timestep)
{
	unsigned int x = blockIdx.x * 3;
	//unsigned int giu = blockDim.x;
	//unsigned int dij = threadIdx.y;
	int stride = 8;

	float3 v1;
	float3 v2;
	float3 v3;

	int idx_1 = indices[x] * stride;
	int idx_2 = indices[x + 1] * stride;
	int idx_3 = indices[x + 2] * stride;

	v1.x = pos[idx_1];
	v1.y = pos[idx_1 + 1];
	v1.z = pos[idx_1 + 2];

	v2.x = pos[idx_2];
	v2.y = pos[idx_2 + 1];
	v2.z = pos[idx_2 + 2];

	v3.x = pos[idx_3];
	v3.y = pos[idx_3 + 1];
	v3.z = pos[idx_3 + 2];

	float small_num = 0.001;
	float3 middle = v1 + v2 + v3;
	v1 = v1 + middle * small_num;
	v2 = v2 + middle * small_num;
	v3 = v3 + middle * small_num;

	pos[idx_1] = v1.x;
	pos[idx_1 + 1] = v1.y;
	pos[idx_1 + 2] = v1.z;

	pos[idx_2] = v2.x;
	pos[idx_2 + 1] = v2.y;
	pos[idx_2 + 2] = v2.z;

	pos[idx_3] = v3.x;
	pos[idx_3 + 1] = v3.y;
	pos[idx_3 + 2] = v3.z;

	//pos[idx_1] = pos[idx_1] + sinf(timestep / 9) / 15.0;
	//pos[idx_1 + 1] = pos[idx_1 + 1] + cosf(timestep / 9) / 15.0;
	//pos[idx_1 + 2] = pos[idx_1 + 2] + cosf(timestep / 9) / 15.0;







	//float mul = 0.9;
	//for (int i = 0; i < 3; i++)
	//{
	//	pos[idx_1] = pos[idx_1] * mul;
	//	pos[idx_2] = pos[idx_2] * mul;
	//	pos[idx_3] = pos[idx_3] * mul;
	//	idx_1 += 1;
	//	idx_2 += 1;
	//	idx_3 += 1;
	//}
}

CudaGLBufferObj::CudaGLBufferObj(void)
{
}

CudaGLBufferObj::~CudaGLBufferObj(void)
{
}

std::string CudaGLBufferObj::getType()
{
	return "CUDA + GL VBO";
}

void CudaGLBufferObj::bind()
{
	//If the parameter size is zero, there is no data to bind to the GL
	if (parameters.size() == 0) {
		std::cout << "Empty buffers for shader object: \n";
		return;
	}

	if (!bound) {
		initBuffers();
		bound = true;
	}
	else
		runCuda();

	glBindVertexArray(vao_handle);
	if (has_indices)
		bindIndices();
}

void CudaGLBufferObj::runCuda()
{
	hipError_t error_code = hipSuccess;
	error_code = hipGraphicsMapResources(1, &cuda_vbo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL buffer" << std::endl;

	size_t num_bytes;
	size_t num_bytes_ibo;
	error_code = hipGraphicsResourceGetMappedPointer(&dev_arg1, &num_bytes, cuda_vbo);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL buffer" << std::endl;

	error_code = hipGraphicsMapResources(1, &cuda_ibo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL index buffer" << std::endl;

	error_code = hipGraphicsResourceGetMappedPointer(&dev_indices, &num_bytes_ibo, cuda_ibo);
	if (error_code != hipSuccess)
		std::cout << "Cuda error mapping GL buffer" << std::endl;

	int timestep;
	launchKernel();
	error_code = hipGraphicsUnmapResources(1, &cuda_vbo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error unmapping GL buffer" << std::endl;
	error_code = hipGraphicsUnmapResources(1, &cuda_ibo, 0);
	if (error_code != hipSuccess)
		std::cout << "Cuda error unmapping GL index buffer" << std::endl;
}

void CudaGLBufferObj::initIndexBuffer(int* indices, int n)
{
	has_indices = true;
	unsigned int* temp = new unsigned int[1];
	num_vertices = n / 4;

	glGenBuffers(1, temp);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, temp[0]);
	index_handle = temp[0];
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, n, indices,
		GL_STATIC_DRAW);
	// Unbind buffer
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

	hipError_t error = hipGraphicsGLRegisterBuffer(&cuda_ibo, index_handle, cudaGraphicsMapFlagsWriteDiscard);

	if (error != hipSuccess)
		std::cout << "Error occurred mapping index buffer to CUDA" << std::endl;

	//Now that the memory is buffered, delete the client-side ints
	delete[] indices;
}

void CudaGLBufferObj::launchKernel()
{
	return;
	dim3 block(num_vertices / 3, 1, 1);
	dim3 threads_per_block(1, 1, 1);
	simple_vbo_kernel<<<block, threads_per_block>>>((float*)dev_arg1, (int*) dev_indices, num_vertices, timestep++);
}

void CudaGLBufferObj::unbind()
{
	glBindVertexArray(0);
}

void CudaGLBufferObj::initBuffers()
{
	timestep = 0;
	//First determine the size of the entire buffer
	int buffer_size = 0;
	//Represents the number of bytes between the ith and i + 1th item in the buffer
	stride = 0;
	for (ShaderParameter p : parameters)
	{
		buffer_size += p.size;
		stride += p.cardinality * 4;
		p.shader_handle = WorldState::getCurrentShader()->getAttrib(p.name);
	}

	int vertex_count = buffer_size / stride;
	float* interleaved_data = new float[buffer_size / 4];
	int k = 0;

	//The following assumes that all parameters have the same number of elements
	for (int i = 0; i < vertex_count; i++)
	{
		for (ShaderParameter p : parameters)
		{
			//The index into the array that will
			int offset = p.cardinality * i;
			for (int j = 0; j < p.cardinality; j++)
				interleaved_data[k++] = (p.client_data[offset + j]);
		}
	}
	//Create a Vertex Array Object so that we can bind the VBOs to it
	glGenVertexArrays(1, &vao_handle);
	glBindVertexArray(vao_handle);
	GLuint* buffer_return = new GLuint[1];
	//Finally generate and bind the buffer
	glGenBuffers(1, buffer_return);
	interleave_handle = buffer_return[0];
	glBindBuffer(GL_ARRAY_BUFFER, interleave_handle);
	glBufferData(GL_ARRAY_BUFFER, buffer_size, &interleaved_data[0], GL_STATIC_DRAW);
	//Describe the layout of the data to the GL
	GLvoid* offset = 0;
	int address = 0;
	for (ShaderParameter parameter : parameters)
	{
		//GLuint shader_num = WorldState::current_shader->getAttrib(parameter.name);
		GLuint shader_num = WorldState::getCurrentShader()->getAttrib(parameter.name);
		if (shader_num == -1)
		{
			address += parameter.cardinality * 4;
			offset = (GLvoid*) address;
			continue;
		}
		glVertexAttribPointer(shader_num, parameter.cardinality, GL_FLOAT, false, stride, offset);
		if (glGetError() != 0)
			std::cout << glGetError() << std::endl << "Illegal arguments set for framebuffer texture";

		address += parameter.cardinality * 4;
		offset = (GLvoid*) address;
		glEnableVertexAttribArray(shader_num);
		if (glGetError() != 0)
			std::cout << glGetError() << std::endl << "Illegal arguments set for framebuffer texture";
	}
	//Unbind
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	hipError_t err = hipGraphicsGLRegisterBuffer(&cuda_vbo, interleave_handle, cudaGraphicsMapFlagsWriteDiscard);
	if (err != 0)
		std::cout << "Could not map GL Buffer to CUDA" << std::endl;

	delete[] buffer_return;
	delete[] interleaved_data;
	//Delete all individual client-side arrays
	for (ShaderParameter p : parameters)
	{
		if (p.delete_on_buffer)
			delete[] p.client_data;
	}
	bound = true;
}

hipGraphicsResource_t CudaGLBufferObj::getVBO()
{
	return cuda_vbo;
}

hipGraphicsResource_t CudaGLBufferObj::getIBO()
{
	return cuda_ibo;
}