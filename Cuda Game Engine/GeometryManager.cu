#include "hip/hip_runtime.h"
#include "GeometryManager.cuh"

__device__ float3 operator+(const float3 &a, const float3 &b)
{
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator* (const float3 &a, const float &b)
{
	return make_float3(a.x * b, a.y * b, a.z * b);
}


__global__ void simple_vbo_kernel(float *pos1, int num_triangles, int timestep)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x > num_triangles)
	{
		x = 2;
		return;
	}
	int stride = 8;

	float3 v1;
	float3 v2;
	float3 v3;

	int idx_1 = 3 * x * stride;
	int idx_2 = (3 * x + 1) * stride;
	int idx_3 = (3 * x + 2) * stride;

	v1.x = pos1[idx_1];
	v1.y = pos1[idx_1 + 1];
	v1.z = pos1[idx_1 + 2];

	v2.x = pos1[idx_2];
	v2.y = pos1[idx_2 + 1];
	v2.z = pos1[idx_2 + 2];

	v3.x = pos1[idx_3];
	v3.y = pos1[idx_3 + 1];
	v3.z = pos1[idx_3 + 2];

	float small_num = 0.0003;
	float3 middle = v1 + v2 + v3;
	v1 = v1 + middle * small_num;
	v2 = v2 + middle * small_num;
	v3 = v3 + middle * small_num;

	pos1[idx_1] = v1.x;
	pos1[idx_1 + 1] = v1.y;
	pos1[idx_1 + 2] = v1.z;

	pos1[idx_2] = v2.x;
	pos1[idx_2 + 1] = v2.y;
	pos1[idx_2 + 2] = v2.z;

	pos1[idx_3] = v3.x;
	pos1[idx_3 + 1] = v3.y;
	pos1[idx_3 + 2] = v3.z;
}

GeometryManager::GeometryManager() :
	initialized(false),
	timestep(0)
{
}

void GeometryManager::processGeometry(std::vector<TriangleMesh>& meshes)
{
	vec3 posn(0.0, 0.0, -5.0);

	if (!initialized)
	{
		if (cuda_buffers.size() < meshes.size())
		{
			for (TriangleMesh& mesh : meshes)
			{
				if (mesh.static_object)
					continue;
				//CudaArg to_add = {mesh.getState().getBufferManager().getVBO(), 0};
				//cuda_buffers.push_back( to_add );
			}
		}
		initialized = true;
	}
	runCuda();
}

void GeometryManager::runCuda()
{
	hipError_t error_code = hipSuccess;
	size_t num_bytes;
	size_t num_bytes_ibo;

	for (CudaArg& arg : cuda_buffers)
	{
		error_code = hipGraphicsMapResources(1, &arg.buffer, 0);
		if (error_code != hipSuccess)
			std::cout << "Cuda error mapping GL buffer" << std::endl;

		error_code = hipGraphicsResourceGetMappedPointer(&arg.dev_arg, &arg.num_bytes, arg.buffer);
		if (error_code != hipSuccess)
			std::cout << "Cuda error mapping GL buffer" << std::endl;
	}

	int timestep;
	launchKernel();

	for (int i = 0; i < cuda_buffers.size(); i++)
	{
		error_code = hipGraphicsUnmapResources(1, &cuda_buffers[i].buffer, 0);
		if (error_code != hipSuccess)
			std::cout << "Cuda error unmapping GL buffer" << std::endl;
	}
}

void GeometryManager::launchKernel()
{
	for (CudaArg& arg : cuda_buffers)
	{
		int num_triangles = arg.num_bytes / 96;
		dim3 block(num_triangles, 1, 1);
		dim3 threads_per_block(1, 1, 1);
		simple_vbo_kernel<<<block, threads_per_block>>>((float*)arg.dev_arg, num_triangles, timestep++);
	}
}


GeometryManager::~GeometryManager()
{
}